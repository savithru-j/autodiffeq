#include "hip/hip_runtime.h"
// autodiffeq - C++ library for sensitivity analysis of ODEs
// Licensed under the MIT License (http://opensource.org/licenses/MIT)
// Copyright (c) 2023, Savithru Jayasinghe

#include <gtest/gtest.h>
#include <autodiffeq/linearalgebra/Array1D.hpp>
#include <autodiffeq/linearalgebra/GPUArray1D.cuh>

#include <complex>

using namespace autodiffeq;

template<typename T = double>
__global__
void add(const DeviceArray1D<T>& x, const DeviceArray1D<T>& y, DeviceArray1D<T>& z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  auto size = x.size();
  if (i < size) 
    z[i] = x[i] + y[i];
}

//----------------------------------------------------------------------------//
TEST( GPUArray1D, Constructor )
{
  {
    GPUArray1D<double> vec(3);
    EXPECT_EQ(vec.size(), 3u);
  }

  {
    GPUArray1D<std::complex<double>> vec(5, {0.25, -1.0});
    EXPECT_EQ(vec.size(), 5u);
    auto vec_h = vec.CopyToHost();
    for (int i = 0; i < 5; ++i)
    {
      EXPECT_EQ(vec_h[i].real(), 0.25);
      EXPECT_EQ(vec_h[i].imag(), -1.0);
    }
  }

  {
    GPUArray1D<int> vec = {-2, 3, 6, 42, -4, 8};
    auto vec_h = vec.CopyToHost();
    EXPECT_EQ(vec.size(), 6u);
    EXPECT_EQ(vec_h.size(), 6u);
    EXPECT_EQ(vec_h[0], -2);
    EXPECT_EQ(vec_h[1], 3);
    EXPECT_EQ(vec_h[2], 6);
    EXPECT_EQ(vec_h[3], 42);
    EXPECT_EQ(vec_h[4], -4);
    EXPECT_EQ(vec_h[5], 8);

    GPUArray1D<int> vec2 = {};
    EXPECT_EQ(vec2.size(), 0u);
  }

  {
    Array1D<int> vec_cpu = {-2, 3, 6, 42, -4, 8};
    GPUArray1D<int> vec(vec_cpu);
    auto vec_h = vec.CopyToHost();
    EXPECT_EQ(vec.size(), 6u);
    EXPECT_EQ(vec_h.size(), 6u);
    EXPECT_EQ(vec_h[0], -2);
    EXPECT_EQ(vec_h[1], 3);
    EXPECT_EQ(vec_h[2], 6);
    EXPECT_EQ(vec_h[3], 42);
    EXPECT_EQ(vec_h[4], -4);
    EXPECT_EQ(vec_h[5], 8);
  }
}

//----------------------------------------------------------------------------//
TEST( GPUArray1D, Add )
{
  int N = 1024;
  GPUArray1D<double> x(N, 5.0);
  GPUArray1D<double> y(N);
  y.SetValue(-2.0);
  GPUArray1D<double> z(N);

  add<<<(N+255)/256, 256>>>(x.GetDeviceArray(), y.GetDeviceArray(), 
                            z.GetDeviceArray());
  
  auto z_h = z.CopyToHost();
  for (int i = 0; i < N; ++i)
    EXPECT_EQ(z_h[i], 3);
}
